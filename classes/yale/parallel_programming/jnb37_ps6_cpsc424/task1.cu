#include "hip/hip_runtime.h"
#ifndef SERIAL
#define SERIAL 0
#endif
#ifndef FP
#define FP float
#endif

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>

__global__ void gpu_rect_matrixmult(FP *a,FP *b, FP *c,
                                    int n, int p, int m) {

  int col = threadIdx.x + blockDim.x * blockIdx.x;
  int row = threadIdx.y + blockDim.y * blockIdx.y;

  int indexb = col;
  
  if(col < m && row < n) {
    int index = row * m + col;

    c[index] = 0.;
    for (int indexa = row*p; indexa < (row*p + p); indexa++)
      for(){
      c[index] += a[indexa]*b[indexb];
    }
  }
}

void cpu_matrixmult(FP *a,FP *b, FP *c, int n) {

  int index, indexa, indexb;
  FP cvalue;
  for(int col=0;col < n; col++)
    for(int row=0;row < n; row++) {
      indexb = col;
      index = row * n + col;
      cvalue = 0.;
      for (indexa = row*n; indexa < (row*n + n); indexa++, indexb+=n) 
        cvalue += a[indexa]*b[indexb];
      c[index] -= cvalue; //NOTE: This calculates the diff between CPU and GPU computations.
    }
}

__global__ void gpu_sqr_matrixmult(FP *a,FP *b, FP *c, int n) {

  int col = threadIdx.x + blockDim.x * blockIdx.x;
  int row = threadIdx.y + blockDim.y * blockIdx.y;

  int indexb = col;
  int index = row * n + col;

  if(col < n && row < n) {
    c[index] = 0.;
    for (int indexa = row*n; indexa < (row*n + n); indexa++, indexb+=n) 
      c[index] += a[indexa]*b[indexb];
  }

}


// implements the kij matmult method
void cpu_kij_matrixmult(FP *a, FP *b, FP *c,
                    int n, int p, int m){
  FP r;
  FP cvalue;

  for(int k = 0; k < p; k++){
    for(int i = 0; i < n; i++){
      // hold element of A constant
      r = a[i*p + k];
      cvalue = 0.;
      // loop through rows of B and C
      for(int j = 0; j < m; j++){
        c[i*m + j] -= r * b[k*m + j];
        //cvalue += r * b[k*m + j];
        //c[i*m + j] -= cvalue;//NOTE: This calculates the diff between CPU and GPU computations.
      }
    }
  }
}

int main(int argc, char *argv[]) {

  int i, j; // loop counters

  int gpucount = 0; // Count of available GPUs
  int gpunum = 0; // Device number to use
  int Grid_X = 1; //Grid dimension, x and y, square
  int Grid_Y = 1; 
  int Block_Dim = 1; //Block dimension, x and y, square

  int m,n,p; // matrix dimension
  FP *a,*b,*c;
  FP *dev_a, *dev_b, *dev_c;
  int sizeA,sizeB,sizeC; // number of bytes in arrays

  hipEvent_t start, stop; // using cuda events to measure time
  float elapsed_time_ms; // which is applicable for asynchronous code also
  hipError_t errorcode;

  // --------------------SET PARAMETERS AND DATA -----------------------

  errorcode = hipGetDeviceCount(&gpucount);
  if (errorcode == hipErrorNoDevice) {
    printf("No GPUs are visible\n");
    exit(-1);
  }
  else {
     printf("Device count = %d\n",gpucount);
  }

  if ((argc<7) || (argc>8)) {
    printf("Usage: matmul  <n> <p> <m> <block dim> <grid dim> [<dev num>]\n");
    exit (-1);
  }

  n = atoi(argv[1]);
  p = atoi(argv[2]);
  m = atoi(argv[3]);

  Block_Dim = atoi(argv[4]); // Square block
  if (Block_Dim*Block_Dim > 1024) {
    printf("Error, too many threads in block\n");
    exit (-1);
  }

  Grid_X = atoi(argv[5]); // Square grid

  Grid_Y = atoi(argv[6]); // Square grid

  if (argc==5) {
    gpunum = atoi(argv[7]); // Device number
    if ((gpunum > 2) || (gpunum < 0)) {
      printf("Error, Device number must be 0, 1, or 2\n");
      exit (-1);
    }
  }
  Grid_X = m / Block_Dim;
  Grid_Y = n / Block_Dim;

  printf("Block_Dim = %d, Grid_X = %d Grid_Y = %d\n",Block_Dim,Grid_X,Grid_Y);
  printf("Matrix Dimensions:\n\t n: %d p: %d m: %d = %d\n",n,p,m,n*m);

  if (Grid_X*Block_Dim < m || Grid_Y * Block_Dim < n) {
    printf("Error, number of threads in x/y dimensions less than number of array elements\n");
    exit (-1);
  }
  hipSetDevice(gpunum);
  printf("Using device %d\n",gpunum);
  

  dim3 Grid(Grid_X, Grid_Y); //Grid structure
  dim3 Block(Block_Dim, Block_Dim); //Block structure

  sizeA = n * p * sizeof(FP); // number of bytes in total in arrays
  sizeB = p * m * sizeof(FP);
  sizeC = n * m * sizeof(FP);

  a = (FP*) malloc(sizeA); // dynamically allocated memory for arrays on host
  b = (FP*) malloc(sizeB);
  c = (FP*) malloc(sizeC); // results from GPU

  srand(12345);
  for(i=0;i < n;i++)
    for(j=0;j < p;j++) {
      a[i * p + j] = (FP) rand() / (FP) RAND_MAX;
      //      a[i * p + j] = (FP) i+j; // may be helpful for debugging
    }

  for(i=0;i < p;i++)
    for(j=0;j < m;j++) {
      b[i * n + j] = (FP) rand() / (FP) RAND_MAX;
      //      b[i * n + j] = (FP) i+j; // may be helpful for debugging
    }

  // ------------- COMPUTATION DONE ON GPU ----------------------------

  hipMalloc((void**)&dev_a, sizeA); // allocate memory on device
  hipMalloc((void**)&dev_b, sizeB);
  hipMalloc((void**)&dev_c, sizeC);

  hipMemcpy(dev_a, a , sizeA ,hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b , sizeB ,hipMemcpyHostToDevice);

  hipEventCreate(&start); // instrument code to measure start time
  hipEventCreate(&stop);
  
  hipEventRecord(start, 0);
  // hipEventSynchronize(start); // not needed

  gpu_rect_matrixmult<<<Grid,Block>>>(dev_a,dev_b,dev_c,n,p,m);

  hipEventRecord(stop, 0); // instrument code to measure end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  hipMemcpy(c,dev_c, sizeC ,hipMemcpyDeviceToHost);

  printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms); // exec. time

  if(SERIAL == 1){

  // ------------- COMPUTATION DONE ON HOST CPU ----------------------------
  // DEBUGGING USE ONLY (AND FOR LIMITED NUMBERS OF TIMING RUNS)

  hipEventRecord(start, 0); // use same timing
  // hipEventSynchronize(start); // not needed


  cpu_kij_matrixmult(a,b,c, n,p,m); // do calculation on host (NOTE: This computes the diff with GPU result.)

  hipEventRecord(stop, 0); // instrument code to measue end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  printf("Time to calculate results on CPU: %f ms.\n", elapsed_time_ms); // exec. time

// ------------------- check device creates correct results -----------------

  double error, suma, sumb, sumc, ai, bi, ci;
  suma = 0.; sumb = 0; sumc = 0;
  for(i=0;i < n*m;i++) {
    ai = (double) a[i];
    bi = (double) b[i];
    ci = (double) c[i];
    suma += ai*ai;
    sumb += bi*bi;
    sumc += ci*ci;
  }
  suma = sqrt(suma);
  sumb = sqrt(sumb);
  sumc = sqrt(sumc);
  error =  sumc/(sqrtf(n*m)*suma*sumb);
  printf("Scaled error between GPU and CPU: %e\n", error);

  }
// -------------- clean up ---------------------------------------

  free(a);
  free(b);
  free(c);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;

  }
