#ifndef SERIAL
#define SERIAL 0
#endif
#ifdef DEBUG
    #define DEBUG_PRINT printf
#else
#define DEBUG_PRINT 
#endif
#ifndef FP
#define FP float
#endif

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>



__global__ void gpu_rect_matrixmult(FP *a,FP *b, FP *c,
                                    int n, int p, int m) {

  int col = threadIdx.x + blockDim.x * blockIdx.x;
  int row = threadIdx.y + blockDim.y * blockIdx.y;

  int indexb = col;
  int index = row * m + col;

  if(col < m && row < n) {
    c[index] = 0.;
    for (int indexa = row*p; indexa < (row*p + p); indexa++, indexb+=m){
      c[index] += a[indexa]*b[indexb];
    }
  }
}


__global__ void tiled_matrixmult(FP *a,FP *b, FP *c,
                                 int n, int p, int m,
                                 int tileDim) {

  int col = threadIdx.x + blockDim.x * blockIdx.x;
  int row = threadIdx.y + blockDim.y * blockIdx.y;

  int index = threadIdx.y*tileDim + threadIdx.x;
  int indexC = blockIdx.x * tileDim + blockIdx.y * tileDim * m + threadIdx.y*m + threadIdx.x;
  c[indexC] = 0.;


  // dynamically allocate shared memory
  extern __shared__ FP buffer[];
  //divide shared buffer between sub matrix A and B
  FP *Asub = &buffer[0];
  FP *Bsub = &buffer[tileDim*tileDim];


  // Shift tile across matrixes
  for(int i = 0; i < p /(float)  tileDim; i++ ){

    int widthOverflowA = (i+1)*tileDim;

    // where are the submatrices in the orignal matricies?
    int indexA = i*tileDim + blockIdx.y*tileDim*p + threadIdx.y*p + threadIdx.x;
    int indexB = blockIdx.x*tileDim + i*tileDim*m + threadIdx.y*m + threadIdx.x;


    //make sure to sync after computation!!!


    if(col < m && row < n){

      // We only want to calculate threads that are
      // in each matrix
      if(i*tileDim + threadIdx.x< p){
        Asub[index] = a[indexA];
        //DEBUG_PRINT("c[%d] a[%d] = %f at Asub[%d]\n",indexC,indexA,a[indexA],index);
      }
      else{
        
        Asub[index] = 0;
      }
      if( i*tileDim + threadIdx.y < p){
        Bsub[index] = b[indexB];
        //DEBUG_PRINT("c[%d] b[%d] = %f at Bsub[%d]\n",indexC,indexB,b[indexB],index);

      }
      else{
        Bsub[index] = 0;
      }



        /* if(blockIdx.x == 0 && blockIdx.y == 0){ */
        /*   DEBUG_PRINT("ITERATION: %d\n",i); */
        /*   DEBUG_PRINT("index: %d\n", index); */
        /*   DEBUG_PRINT("row: %d col: %d\n",row,col); */
        /*   //DEBUG_PRINT("c[%d] Asub[%d] = a[%d] = %f Bsub[%d] = b[%d] = %f\n",indexC,index,indexA,a[indexA],index,indexB,b[indexB]); */
        /* }  */



      __syncthreads();

      int bound  =  widthOverflowA  > p ? p % tileDim: tileDim;
      DEBUG_PRINT("id: %d i: %d  overflow: %d Bound: %d\n",blockIdx.x,i,widthOverflowA,bound);

      for(int iA = threadIdx.y*tileDim ,iB = threadIdx.x ; iA <(threadIdx.y*tileDim + bound); iA++, iB+=tileDim){
        c[indexC] += Asub[iA] * Bsub[iB];
        if(blockIdx.x == 0 && blockIdx.y == 0 ){
          DEBUG_PRINT(" c[%d] = Asub[%d]:%f * Bsub[%d]:%f = %f\n",indexC,iA,Asub[iA],iB,Bsub[iB],c[indexC]);
          DEBUG_PRINT("c val: %f\n",c[indexC]);
        }
      }
      DEBUG_PRINT("c[%d] = %f\n",indexC, c[indexC]);
      __syncthreads();
    }
  }
}

void cpu_matrixmult(FP *a,FP *b, FP *c, int n) {

  int index, indexa, indexb;
  FP cvalue;
  for(int col=0;col < n; col++)
    for(int row=0;row < n; row++) {
      indexb = col;
      index = row * n + col;
      cvalue = 0.;
      for (indexa = row*n; indexa < (row*n + n); indexa++, indexb+=n)
        cvalue += a[indexa]*b[indexb];
      c[index] -= cvalue; //NOTE: This calculates the diff between CPU and GPU computations.
    }
}


// implements the kij matmult method
void cpu_kij_matrixmult(FP *a, FP *b, FP *c,
                        int n, int p, int m){
  FP r;

  for(int k = 0; k < p; k++){
    for(int i = 0; i < n; i++){
      // hold element of A constant
      r = a[i*p + k];
      // loop through rows of B and C
      for(int j = 0; j < m; j++){
        c[i*m + j] -= r * b[k*m + j];
        //cvalue += r * b[k*m + j];
        //c[i*m + j] -= cvalue;//NOTE: This calculates the diff between CPU and GPU computations.
      }
    }
  }
}

int main(int argc, char *argv[]) {

  int i, j; // loop counters

  int gpucount = 0; // Count of available GPUs
  int gpunum = 0; // Device number to use
  int Block_Dim = 1;
  int Grid_X = 1; //Grid dimension, x and y, square
  int Grid_Y = 1; 

  int m,n,p; // matrix dimension
  FP *a,*b,*c;
  FP *dev_a, *dev_b, *dev_c;
  int sizeA,sizeB,sizeC; // number of bytes in arrays

  hipEvent_t start, stop; // using cuda events to measure time
  float elapsed_time_ms; // which is applicable for asynchronous code also
  hipError_t errorcode;

  // --------------------SET PARAMETERS AND DATA -----------------------

  errorcode = hipGetDeviceCount(&gpucount);
  if (errorcode == hipErrorNoDevice) {
    printf("No GPUs are visible\n");
    exit(-1);
  }
  else {
     printf("Device count = %d\n",gpucount);
  }

  if ((argc<7) || (argc>8)) {
    printf("Usage: matmul  <n> <p> <m> <block dim> <grid dim> [<dev num>]\n");
    exit (-1);
  }

  n = atoi(argv[1]);
  p = atoi(argv[2]);
  m = atoi(argv[3]);

  Block_Dim = atoi(argv[4]); // Square block
  if (Block_Dim*Block_Dim > 1024) {
    printf("Error, too many threads in block\n");
    exit (-1);
  }

  //get Grid dims from input
  Grid_X = atoi(argv[5]); // Square grid
  Grid_Y = atoi(argv[6]); // Square grid

  if (argc==6) {
    gpunum = atoi(argv[7]); // Device number
    if ((gpunum > 2) || (gpunum < 0)) {
      printf("Error, Device number must be 0, 1, or 2\n");
      exit (-1);
    }
  }

  // even though we get them from
  // input its easier and more efficent to
  // do this
  Grid_X = m / Block_Dim;
  Grid_Y = n / Block_Dim;

  printf("Block_Dim = %d, Grid_X = %d Grid_Y = %d\n",Block_Dim,Grid_X,Grid_Y);
  printf("Matrix Dimensions:\n\t n: %d p: %d m: %d = %d\n",n,p,m,n*m);

  if (Grid_X*Block_Dim < m || Grid_Y * Block_Dim < n) {
    printf("Error, number of threads in x/y dimensions less than number of array elements\n");
    exit (-1);
  }
  hipSetDevice(gpunum);
  printf("Using device %d\n",gpunum);
  

  dim3 Grid(Grid_X, Grid_Y); //Grid structure
  dim3 Block(Block_Dim, Block_Dim); //Block structure

  sizeA = n * p * sizeof(FP); // number of bytes in total in arrays
  sizeB = p * m * sizeof(FP);
  sizeC = n * m * sizeof(FP);

  a = (FP*) malloc(sizeA); // dynamically allocated memory for arrays on host
  b = (FP*) malloc(sizeB);
  c = (FP*) malloc(sizeC); // results from GPU

  srand(12345);
  DEBUG_PRINT("Matrix A:\n");
  for(i=0;i < n;i++)
    for(j=0;j < p;j++) {
      a[i * p + j] = (FP) rand() / (FP) RAND_MAX;
      //a[i * p + j] = (FP) i+j; // may be helpful for debugging
      DEBUG_PRINT("%f ",a[i*p+j]);
    }

  DEBUG_PRINT("\nMatrix B:\n");
  for(i=0;i < p;i++)
    for(j=0;j < m;j++) {
      b[i * n + j] = (FP) rand() / (FP) RAND_MAX;
      //b[i * m + j] = (FP) i+j; // may be helpful for debugging
      DEBUG_PRINT("%f ",b[i*m+j]);
    }

  // ------------- COMPUTATION DONE ON GPU ----------------------------

  hipMalloc((void**)&dev_a, sizeA); // allocate memory on device
  hipMalloc((void**)&dev_b, sizeB);
  hipMalloc((void**)&dev_c, sizeC);


  hipMemcpy(dev_a, a , sizeA ,hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b , sizeB ,hipMemcpyHostToDevice);

  hipEventCreate(&start); // instrument code to measure start time
  hipEventCreate(&stop);

  hipEventRecord(start, 0);
  // cudaEventSynchronize(start); // not needed

  size_t tileDim = Block_Dim*Block_Dim*2;
  //dynamically allocate memory
  tiled_matrixmult<<<Grid,Block,tileDim*sizeof(FP)>>>(dev_a,dev_b,dev_c,n,p,m,Block_Dim);

  hipEventRecord(stop, 0); // instrument code to measure end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  hipMemcpy(c,dev_c, sizeC ,hipMemcpyDeviceToHost);

  //for debugging
  for(int i = 0; i < n; i++){
    DEBUG_PRINT("\n");
    for(int j = 0; j < m; j++)
      DEBUG_PRINT("%f ",c[i*m + j]);
  }

  printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms); // exec. time

  if(SERIAL == 1){
    
  // ------------- COMPUTATION DONE ON HOST CPU ----------------------------
  // DEBUGGING USE ONLY (AND FOR LIMITED NUMBERS OF TIMING RUNS)

  hipEventRecord(start, 0); // use same timing
  // cudaEventSynchronize(start); // not needed


  cpu_kij_matrixmult(a,b,c, n,p,m); // do calculation on host (NOTE: This computes the diff with GPU result.)

  hipEventRecord(stop, 0); // instrument code to measue end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  printf("Time to calculate results on CPU: %f ms.\n", elapsed_time_ms); // exec. time

// ------------------- check device creates correct results -----------------


  double error, suma, sumb, sumc, ai, bi, ci;
  suma = 0.; sumb = 0; sumc = 0;
  for(i=0;i < n*m;i++) {
    ai = (double) a[i];
    bi = (double) b[i];
    ci = (double) c[i];
    suma += ai*ai;
    sumb += bi*bi;
    sumc += ci*ci;
  }
  suma = sqrt(suma);
  sumb = sqrt(sumb);
  sumc = sqrt(sumc);
  error =  sumc/(sqrtf(n*m)*suma*sumb);
  printf("Scaled error between GPU and CPU: %e\n", error);
  }

// -------------- clean up ---------------------------------------

  free(a);
  free(b);
  free(c);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
